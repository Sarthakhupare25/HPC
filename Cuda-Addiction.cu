

#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void add(int *A, int *B, int *C, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < size){
        C[tid] = A[tid] +B[tid];
    }
}

void initialize(int *vector, int n){
    for(int i =0;i < n; i++){
        vector[i] = rand() % 20 +1;
    }
}

void print(int *vector, int size){
    for(int i = 0; i<size;i++){
        cout<<vector[i]<<" ";
    }
    cout<<endl;
}

int main(){
    int *A, *B, *C;
    int n = 4;
    int vectorSize = n;
    int vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, n);
    initialize(B, n);

    cout<<"A:\n";
    print(A, n);
    cout<<"B:\n";
    print(B, n);

    int *X, *Y, *Z;

    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadperBlock = 256;
    int blocksperGrid = (n + threadperBlock - 1)/ threadperBlock;

    add<<<blocksperGrid, threadperBlock>>>(X, Y, Z, n);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout<<"C:\n";
    print(C, n); 
    

    delete[] A;
    delete[] B;
    delete[] C;


    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;

}
// Commands to run
// nvcc filename.cu
// ./a.out